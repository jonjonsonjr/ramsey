#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "Fitness.h"

const int threadsPerBlock = 32*32;
const int blocksPerGrid = 32*32;

__device__ __constant__ int choose_cache[44][6];
__device__ __constant__ char adj[43][43];

__global__ void eval(int *c) 
{
    __shared__ short cache[threadsPerBlock];
    int cacheIndex = threadIdx.x;
	int sum = 0; // number of cliques found by this thread
	int arr[5] = { 0, 0, 0, 0, 0 };

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = gridDim.x * blockDim.x; // our "stride" for multiple passes

	while (tid < UPPER_BOUND) {
		int a = N;
		int b = K;
		int x = 962597 - tid; // x is the "dual" of m

		for (int i = 0; i < K; i++) {
			int v = a - 1;

			while (choose_cache[v][b] > x) {
				v--;
			}

			arr[i] = v;
			x = x - choose_cache[arr[i]][b];
			a = arr[i];
			b--;
		}

		for (int i = 0; i < K; i++) {
			arr[i] = (N - 1) - arr[i];
		}
            
		int result = adj[arr[0]][arr[1]] +
					 adj[arr[0]][arr[2]] +
					 adj[arr[0]][arr[3]] +
					 adj[arr[0]][arr[4]] +
					 adj[arr[1]][arr[2]] +
					 adj[arr[1]][arr[3]] +
					 adj[arr[1]][arr[4]] +
					 adj[arr[2]][arr[3]] +
					 adj[arr[2]][arr[4]] +
					 adj[arr[3]][arr[4]];
            
		sum += (result == 0 || result == KC2);

		tid += offset; // move on to next pass
	}

    cache[cacheIndex] = sum; // populate the cache values
    __syncthreads(); // synchronize threads in this block

    // threadsPerBlock must be a power of 2 because of the following:
    int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}

		__syncthreads(); // be sure every array has finished
		i /= 2;
	}

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0]; //number of cliques found by this block
	}
}

int CudaEval(char *adjacency_matrix)
{
	int           c, *partial_c;
    int           *dev_partial_c;

	partial_c = (int*) malloc(blocksPerGrid * sizeof(int));
	hipMalloc((void**) &dev_partial_c, blocksPerGrid * sizeof(int));

	hipMemcpyToSymbol(HIP_SYMBOL(adj), adjacency_matrix, 43 * 43 * sizeof(char));

	eval<<<blocksPerGrid,threadsPerBlock>>>(dev_partial_c);

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

	// sum all the blocks' sums
	c = 0;

	for (int i = 0; i < blocksPerGrid; i++) {
		c = c + partial_c[i];
	}

	hipFree(dev_partial_c);
	free(partial_c);
	
	return c;
}

void CudaInit()
{
	int h_choose_cache[][6] = {
        {0, 0, 0, 0, 0, 0},
        {0, 1, 0, 0, 0, 0},
        {0, 2, 1, 0, 0, 0},
        {0, 3, 3, 1, 0, 0},
        {0, 4, 6, 4, 1, 0},
        {0, 5, 10, 10, 5, 1},
        {0, 6, 15, 20, 15, 6},
        {0, 7, 21, 35, 35, 21},
        {0, 8, 28, 56, 70, 56},
        {0, 9, 36, 84, 126, 126},
        {0, 10, 45, 120, 210, 252},
        {0, 11, 55, 165, 330, 462},
        {0, 12, 66, 220, 495, 792},
        {0, 13, 78, 286, 715, 1287},
        {0, 14, 91, 364, 1001, 2002},
        {0, 15, 105, 455, 1365, 3003},
        {0, 16, 120, 560, 1820, 4368},
        {0, 17, 136, 680, 2380, 6188},
        {0, 18, 153, 816, 3060, 8568},
        {0, 19, 171, 969, 3876, 11628},
        {0, 20, 190, 1140, 4845, 15504},
        {0, 21, 210, 1330, 5985, 20349},
        {0, 22, 231, 1540, 7315, 26334},
        {0, 23, 253, 1771, 8855, 33649},
        {0, 24, 276, 2024, 10626, 42504},
        {0, 25, 300, 2300, 12650, 53130},
        {0, 26, 325, 2600, 14950, 65780},
        {0, 27, 351, 2925, 17550, 80730},
        {0, 28, 378, 3276, 20475, 98280},
        {0, 29, 406, 3654, 23751, 118755},
        {0, 30, 435, 4060, 27405, 142506},
        {0, 31, 465, 4495, 31465, 169911},
        {0, 32, 496, 4960, 35960, 201376},
        {0, 33, 528, 5456, 40920, 237336},
        {0, 34, 561, 5984, 46376, 278256},
        {0, 35, 595, 6545, 52360, 324632},
        {0, 36, 630, 7140, 58905, 376992},
        {0, 37, 666, 7770, 66045, 435897},
        {0, 38, 703, 8436, 73815, 501942},
        {0, 0, 741, 9139, 82251, 575757},
        {0, 0, 0, 9880, 91390, 658008},
        {0, 0, 0, 0, 101270, 749398},
        {0, 0, 0, 0, 0, 850668},
        {0, 0, 0, 0, 0, 962598}};

	hipMemcpyToSymbol(HIP_SYMBOL(choose_cache), h_choose_cache, sizeof(int) * 44 * 6);
}